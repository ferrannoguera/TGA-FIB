#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <vector>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
using namespace std;

unsigned int total_points, total_values, K, max_iterations;

#define THREADS 1024
#define SIZE 16 
//#define POINT_DIM 2    

__global__ void updateCentroids(double *PointValues, double *KCentroids,
 int *updatingK, int *indexK, int total_points, int total_values, int K){
	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = i * total_points;
	if (i<K) {
		int indcopy = ind;
		for (int j = 0; j<indexK[i]; j++, indcopy++){
			for (int k = 0; k<total_values; ++k) {
				KCentroids[i * total_values + k] += PointValues[updatingK[indcopy] * total_values + k];
			}
		}
	}
	__syncthreads();
	if (i<K) {
			for (int k = 0; k<total_values; ++k) {
				KCentroids[i * total_values + k] /= indexK[i];
			}
	}
			
}

/*__global__ void UDist2(int dim, int nk, int np, double *DK, double *TV, double *KV){
    __shared__ double sTV[SIZE * POINT_DIM];//fila completa, una K entera y un P entero x SIZE
    __shared__ double sKV[SIZE * POINT_DIM];
    
    int bx = blockIdx.x; int tx = threadIdx.x;
    int by = blockIdx.y; int ty = threadIdx.y;
    //los #SIZE threads q van a usar la K[N] y la P[M] cargan una parte de ambos, concretamente dim/SIZE valores +1 si no multiplo
    int row = bx * SIZE + tx;
    int col = by * SIZE + ty;
    int indaux = dim/SIZE;
    //carga paralela de sTV y sKV, t*indaux = particion q le toca
    for(int l= 0; l<indaux; l++){
        sTV[tx*dim+ty*indaux+l] = TV[row*dim+ty*indaux+l];
        sKV[ty*dim+tx*indaux+l] = KV[col*dim+ty*indaux+l];
    }
    //carga de las partes no multiplo
    int check = dim%SIZE;
    if(check > 0){
        int actual = tx-ty;
        actual = actual < 0 ? -actual : actual;
        if(actual < check){
            sTV[(tx+1)*dim-check+actual-1] = TV[(row+1)*dim-check+actual-1];//actual-1-check = pos del modulo, siempre q pasemos a fila siguiente
            sKV[(ty+1)*dim-check+actual-1] = KV[(col+1)*dim-check+actual-1];
        }
    }
    __syncthreads();
    //calculo
    if(row < nk && col < np){
        double tmp = 0.0;
        for(int k = 0; k<dim; k++){
            double aux = KV[row*dim+k] - TV[col*dim+k];
            tmp += aux*aux;
        }
        DK[row*np+col] = sqrt(tmp);
    }
}*/


__global__ void UDist(int dim, int nk, int np, double *DK, double *TV, double *KV){
    int col = blockIdx.y * blockDim.y + threadIdx.y;//inv
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(row < nk && col < np){
	//if(row >= 20 & row < 30)printf("row: %d col: %d\n",row,col);
        double tmp = 0.0;
        double aux;
        int ind1 = row*dim;
        int ind2 = col*dim;
        for(int k = 0; k<dim; k++){
            aux = KV[ind1+k] - TV[ind2+k];
            tmp += aux*aux;
        }
	
        DK[row*np+col] = sqrt(tmp);
    }
}

__global__ void Kernel04(double *DK, int *Ind, int *gInd, double *gBD, int tdk) { //numelem es el numero de threads// optimo = numde k

	extern __shared__ double sDKcindexed[];//double
  // Cada thread carga 1 elemento desde la memoria global
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i < tdk){
    sDKcindexed[tid] = DK[i];
    sDKcindexed[tdk+tid] = __int2double_rn(Ind[i]);
  }
  __syncthreads();

  // Hacemos la reduccion en la memoria compartida
  // El thread 0 de cada bloque se encarga de realizar la
  // reduccion en la ultima posicion del bloque si este es impar
  int s = blockDim.x*(blockIdx.x+1) <= tdk ? blockDim.x/2 : (tdk%blockDim.x)/2;//tdk es menor q el nº threas lanzados x definicion
  for (s; s>0; s>>=1) {
    
    if(tid == 0){
        if(s%2==1){
            if(sDKcindexed[0]>sDKcindexed[2*s+1]){
                sDKcindexed[0] = sDKcindexed[2*s+1];
                sDKcindexed[tdk] = sDKcindexed[2*s+1+tdk];
            }
        }
    }
    if (tid < s){
        if(sDKcindexed[tid]>sDKcindexed[tid+s]){
            sDKcindexed[tid] = sDKcindexed[tid + s];
            sDKcindexed[tid+tdk] = sDKcindexed[tid +s +tdk];
        }
    }
    __syncthreads();
  }
  // El thread 0 escribe el resultado de este bloque en la memoria global
  if (tid == 0){
      gBD[blockIdx.x] = sDKcindexed[0];
      gInd[blockIdx.x] = __double2int_rn(sDKcindexed[tdk]);
  }
}




void printClusters(double *PointValues, double *KCentroids, 
									 int *ClusteringValues);
									 
									 
bool updatePointDistances();

int main(int argc, char** argv) {

  unsigned int numBytesPointValues, numBytesKCentroids, 
			 numBytesClustering, numBytesupdatingK, numBytesIndexK, numBytesDistMatrix;
							 
  unsigned int nBlocksFil, nThreadsC, nThreadsYeray, nBlocksCol, nBlocksYeray;
 
  hipEvent_t Y0, Y1, Y2, Y3;
  
  float TiempoUpdateCentroids, TiempoUpdatePointDistances;

  double *h_PointValues, *h_KCentroids, *h_DistMatrix; 

  int *h_ClusteringValues, *h_indexK, *h_updatingK;

  
  double *d_PointValues, *d_KCentroids,  *d_DistMatrix; 
  
  int *d_ClusteringValues, *d_indexK, *d_updatingK;

  
  cin >> total_points >> total_values >> K >> max_iterations;
  
  if(K > total_points)
		cout << "INPUT ERROR: K CANT BE BIGGER THAN TOTAL POINTS" << endl;

	//Reservamos el expacio que necesitaremos en memoria
  numBytesKCentroids = K * total_values * sizeof(double);
  
  numBytesPointValues = total_points * total_values * sizeof(double);
  
  numBytesClustering = total_points * sizeof(int);
  
  numBytesupdatingK = K * total_points * sizeof(int);
  
  numBytesDistMatrix =  K * total_points * sizeof(double);
  
  numBytesIndexK = K * sizeof(int);
  

	//Declaramos los eventos
  hipEventCreate(&Y0);
  
  hipEventCreate(&Y1);
  
  hipEventCreate(&Y2);
  
  hipEventCreate(&Y3);


  // Obtener Memoria en el host
  h_PointValues = (double*) malloc(numBytesPointValues); 
  
  h_KCentroids = (double*) malloc(numBytesKCentroids); 
  
  h_ClusteringValues = (int*) malloc(numBytesClustering);
  
  h_updatingK = (int*) malloc(numBytesupdatingK); 
  
  h_indexK = (int*) malloc(numBytesIndexK);
  
  h_DistMatrix = (double*) malloc(numBytesDistMatrix);


			
	//Lectura de los valores
	for(int i = 0; i < total_points; i++) {

		for(int j = 0; j < total_values; j++) {
			double value;
			cin >> value;
			int ind = i * total_values + j;
			h_PointValues[ind] = value;
		}
		
	}
	
	
	for (int i = 0; i<total_points; ++i) {
		h_ClusteringValues[i] = 0;
	}
	
	vector<int> prohibited_indexes;
	
	srand(1);
	for(int i = 0; i < K; i++) {
		while(true)
		{
			int index_point = rand() % total_points;

			if(find(prohibited_indexes.begin(), prohibited_indexes.end(),
					index_point) == prohibited_indexes.end())
			{
				prohibited_indexes.push_back(index_point);
				h_ClusteringValues[index_point] = i;
				break;
			}
		}
	}
	
	for (int i = 0; i<K; ++i) h_indexK[i] = 0;
	
	for (int i = 0; i<total_points; ++i) {
		int ind = h_ClusteringValues[i] * (total_points) + h_indexK[h_ClusteringValues[i]];
		h_updatingK[ind] = i;
		h_indexK[h_ClusteringValues[i]] = 1+h_indexK[h_ClusteringValues[i]];
	}
	
	for (int a = 0; a<K; ++a) {
		for (int b = 0; b<total_values; ++b) {
			h_KCentroids[a * total_values + b] = 0;
		}
	}	


	
	// Obtener Memoria en el device
	hipMalloc((double**)&d_PointValues, numBytesPointValues); 
	
	hipMalloc((double**)&d_KCentroids, numBytesKCentroids); 
	
	hipMalloc((int**)&d_updatingK, numBytesupdatingK);
	
	hipMalloc((int**)&d_indexK, numBytesIndexK); 
	
	hipMalloc((int**)&d_ClusteringValues, numBytesClustering); 
		
	
	
	// Copiar datos desde el host en el device 
	hipMemcpy(d_PointValues, h_PointValues, numBytesPointValues, 
				hipMemcpyHostToDevice);
	
	hipMemcpy(d_KCentroids, h_KCentroids, numBytesKCentroids, 
				hipMemcpyHostToDevice);
				
	hipMemcpy(d_updatingK, h_updatingK, 
				numBytesupdatingK, hipMemcpyHostToDevice);			
	
	hipMemcpy(d_indexK, h_indexK, 
				numBytesIndexK, hipMemcpyHostToDevice);	
	
	

	// Ejecutar el kernel 
	
	nThreadsC = THREADS;
	nBlocksFil = (K + nThreadsC - 1)/nThreadsC; 
	nBlocksCol = (total_values + nThreadsC - 1)/nThreadsC;

	cout << "nBlocksC: " << nBlocksFil << endl;
	cout << "total_values: " << total_values << endl;
	cout << "nThreadsC: " << nThreadsC << endl;
	

	dim3 dimGridC(1, nBlocksFil, 1);
	dim3 dimBlockC(1, nThreadsC, 1);
	
	printf("\n");
	printf("Kernel UpdateCentroids\n");
	printf("Dimension Block: %d x %d x %d (%d) threads\n", dimBlockC.x, dimBlockC.y, dimBlockC.z, dimBlockC.x * dimBlockC.y * dimBlockC.z);
	printf("Dimension Grid: %d x %d x %d (%d) blocks\n", dimGridC.x, dimGridC.y, dimGridC.z, dimGridC.x * dimGridC.y * dimGridC.z);
  
  
	hipEventRecord(Y0, 0);
	hipEventSynchronize(Y0);
	updateCentroids<<<dimGridC,dimBlockC>>>(d_PointValues, d_KCentroids, d_updatingK, d_indexK, total_points, total_values, K); 
	hipEventRecord(Y1, 0);
	hipEventSynchronize(Y1);
	
	hipDeviceSynchronize();
	

	hipMemcpy(h_KCentroids, d_KCentroids, numBytesKCentroids,
								hipMemcpyDeviceToHost);
	
	
	//Yeray thoughts
	hipMalloc((int**)&d_DistMatrix, numBytesDistMatrix); 
	
	
	// Copiar datos desde el host en el device 
	hipMemcpy(d_PointValues, h_PointValues, numBytesPointValues, 
				hipMemcpyHostToDevice);
	
	hipMemcpy(d_KCentroids, h_KCentroids, numBytesKCentroids, 
				hipMemcpyHostToDevice);
				
	hipMemcpy(d_DistMatrix, h_DistMatrix, 
				numBytesDistMatrix, hipMemcpyHostToDevice);			
	
	
	nThreadsYeray = 16;
    nBlocksYeray = 16;
    
    dim3 dimGridY((K+nThreadsYeray-1)/nThreadsYeray, (total_points+nThreadsYeray-1)/nThreadsYeray+1, 1);
	dim3 dimBlockY(nThreadsYeray, nThreadsYeray, 1);
	
	printf("\n");
	printf("Kernel UDist2\n");
	printf("Dimension Block: %d x %d x %d (%d) threads\n", dimBlockY.x, dimBlockY.y, dimBlockY.z, dimBlockY.x * dimBlockY.y * dimBlockY.z);
	printf("Dimension Grid: %d x %d x %d (%d) blocks\n", dimGridY.x, dimGridY.y, dimGridY.z, dimGridY.x * dimGridY.y * dimGridY.z);
	
	hipEventRecord(Y2, 0);
	hipEventSynchronize(Y2);
	UDist<<<dimGridY,dimBlockY>>>(total_values, K, total_points, d_DistMatrix, d_PointValues, d_KCentroids);
	hipEventRecord(Y3, 0);
	hipEventSynchronize(Y3);
	
	hipDeviceSynchronize();
	
	
	hipMemcpy(h_DistMatrix, d_DistMatrix, numBytesDistMatrix,
								hipMemcpyDeviceToHost);
	
	
	/*cout << endl << endl << endl << "RESULTADO FINAL" << endl;
	printClusters(h_PointValues, h_KCentroids, h_ClusteringValues);*/
	
    ///calculo de nuevo vecindario

	
    //numblocks es la dim.x del grid
	int gridtastic = K/16;
	gridtastic += K%16 == 0 ? 0 : 1;
	dim3 dimGridY2(gridtastic, 1, 1);
	dim3 dimBlockY2(16, 1, 1);
	
    int *h_indexaux = (int*) malloc(K*sizeof(int));
    //double *h_distres = (double*) malloc(gridtastic*sizeof(double));//THREADS/16 => numblocks
    //int *h_indexres = (int*) malloc(gridtastic*sizeof(int));
    double *h_aux = (double*) malloc(K*sizeof(double));
	
    hipMalloc((double**)&d_distres, gridtastic*sizeof(double)); 
	hipMalloc((int**)&d_indexres, gridtastic*sizeof(int));
	hipMalloc((double**)&d_aux, K*sizeof(double));
	hipMalloc((double**)&d_indexaux, K*sizeof(double));
	
	bool ferran = true;
	while(ferran and counter <= max_iterations){
		ferran = true;
		for(int i = 0; i<total_points; i++){
			//1º iter
        
			//carga vector de indices inicial
			for(int l = 0; l<K; l++){
				h_indexaux[l] = l;
			}
			//carga el vector de distancias inicial
			for(int kk = 0; kk<K; kk++){
				h_aux[kk] = h_DistMatrix[kk*total_points+i];
			}
		
			//pasa a device mem
			hipMemcpy(d_aux, h_aux, K*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(d_indexaux, h_indexaux, K*sizeof(double), hipMemcpyHostToDevice);

			//reduccion solo funciona en dim.x
			Kernel04<<<dimGridY2, dimBlockY2, gridtastic*2*sizeof(double)>>>(d_aux, d_indexaux, d_indexres, d_distres, K);
        
			hipMemcpy(h_aux, d_distres, gridtastic*sizeof(double), hipMemcpyDeviceToHost);//grid.x = numblocks = THREADS/16
			hipMemcpy(h_indexaux, d_indexres, (gridtastic*sizeof(int)), hipMemcpyDeviceToHost);
   
			///16 = numtheards.x
			int hf = gridtastic/16;
			hf += gridtastic%16 == 0 ? 0 : 1;
		
			while(hf>1){
				hipMemcpy(d_aux, h_aux, hf*sizeof(double), hipMemcpyHostToDevice);//hf = #result de la redux anterior
				hipMemcpy(d_indexaux, h_indexaux, hf*sizeof(double), hipMemcpyHostToDevice);
				dim3 gridmolona(hf,1,1);
				dim3 blockmolon(16,1,1);
				int sig = hf/16;//16 = num threads;
				sig += hf%16 == 0 ? 0 : 1;
				Kernel04<<<gridmolona, blockmolon, sig*2*sizeof(double)>>>(d_aux, d_indexaux, d_indexres, d_distres, hf);
				hf = sig;
				
				hipMemcpy(h_aux, d_distres, hf*sizeof(double), hipMemcpyDeviceToHost);//grid.x = numblocks
				hipMemcpy(h_indexaux, d_indexres, hf*sizeof(int), hipMemcpyDeviceToHost);

		
			}
			if(ferran & h_ClusteringValues[i] != h_indexaux[0]){
				ferran = false;
			}
			h_ClusteringValues[i] = h_indexaux[0];
		}
	}
  
	//printClusters(h_PointValues, h_KCentroids, h_ClusteringValues);




  

  hipEventElapsedTime(&TiempoUpdateCentroids, Y0, Y1);
  hipEventElapsedTime(&TiempoUpdatePointDistances, Y2, Y3);
  //hipEventElapsedTime(&TiempoTotal,  E1, E5);
  

 
  printf("Tiempo UpdateCentroids function: %4.6f milseg\n", 
		TiempoUpdateCentroids);
  printf("Tiempo UpdatePointDistances function: %4.6f milseg\n", 
		TiempoUpdatePointDistances);
  /*printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);*/
  
    hipEventDestroy(Y0); hipEventDestroy(Y1); hipEventDestroy(Y2);
  hipEventDestroy(Y3); 
  

   //Liberar Memoria del device 
	hipFree(d_PointValues); hipFree(d_KCentroids); 
	hipFree(d_ClusteringValues); hipFree(d_updatingK);
	hipFree(d_indexK); hipFree(d_DistMatrix);


	

	//Liberar memoria del host
  free(h_PointValues); free(h_KCentroids); free(h_ClusteringValues);
  free(h_updatingK); free(h_indexK); free(h_DistMatrix);

}


void printClusters(double *PointValues, double *KCentroids, 
									 int *ClusteringValues) {
										 
	for (int i = 0; i<K; ++i) {
		cout << "Centroid " << i << ": ";
		for (int j = 0; j<total_values; ++j) {
			int ind = i * total_values + j;
			cout << KCentroids[ind] << " ";
		}
		cout << endl;
	}
	/*for (int i = 0; i<total_points; ++i) {
		cout << "Point " << i << ": ";
		for (int j = 0; j<total_values; ++j) {
			int ind = i * total_values + j;
			cout << PointValues[ind] << " ";
		}
		cout << "is located on cluster: " << ClusteringValues[i] << endl;
	}*/
}