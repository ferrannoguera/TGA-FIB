
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <vector>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
using namespace std;

//vector< vector<double> > PointValues; 
//vector< vector<double> > KCentroids;
//vector<int> ClusteringValues;
unsigned int total_points, total_values, K, max_iterations;

#define THREADS 8



__global__ void updateCentroids(double *PointValues, double *KCentroids, 
								double *ClusteringValues, int total_points, int total_values, int K){
	
	int kevaluada = blockIdx.y * blockDim.y + threadIdx.y;
	
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int ind = j;
	
	float tmp = 0.0;
	int count = 0;
	if (j < total_values) {

		for (int i = 0; i<total_points; ++i, ind = ind + total_values) {
			if (kevaluada == ClusteringValues[i]) {
				tmp += PointValues[ind];
				++count;
			}
		}
		KCentroids[kevaluada * j + total_values] = tmp/count;
	}
}

void printClusters(double *PointValues, double *KCentroids, 
									 double *ClusteringValues);
									 
//void updateCentroids(double *PointValues, double *KCentroids, 
//										 double *ClusteringValues);
										 
bool updatePointDistances();

void CheckCudaError(char sms[], int line);

int main(int argc, char** argv) {

  unsigned int numBytesPointValues, numBytesKCentroids, 
							 numBytesClustering;
							 
  unsigned int nBlocksC, nThreadsC;
 
  hipEvent_t E1, E2, E3, E4, E5;
  
  float TiempoTotal, TiempoUpdateCentroids, TiempoUpdatePointDistances;

  double *h_PointValues, *h_KCentroids, *h_ClusteringValues;
  
  double *d_PointValues, *d_KCentroids, *d_ClusteringValues;
  
  cin >> total_points >> total_values >> K >> max_iterations;
  
  if(K > total_points)
		cout << "INPUT ERROR: K CANT BE BIGGER THAN TOTAL POINTS" << endl;

	//Reservamos el expacio que necesitaremos en memoria
  numBytesKCentroids = K * total_values * sizeof(double);
  
  numBytesPointValues = total_points * total_values * sizeof(double);
  
  numBytesClustering = total_points * sizeof(double);
  

	//Declaramos los eventos
  hipEventCreate(&E1);
  
  hipEventCreate(&E2);
  
  hipEventCreate(&E3);
  
  hipEventCreate(&E4);
  
  hipEventCreate(&E5);


  // Obtener Memoria en el host
  h_PointValues = (double*) malloc(numBytesPointValues); 
  
  h_KCentroids = (double*) malloc(numBytesKCentroids); 
  
  h_ClusteringValues = (double*) malloc(numBytesClustering);

			
	//Lectura de los valores
	for(int i = 0; i < total_points; i++) {

		for(int j = 0; j < total_values; j++) {
			double value;
			cin >> value;
			int ind = i * total_values + j;
			h_PointValues[ind] = value;
		}
		
	}
	vector<int> prohibited_indexes;
	srand(1);
	for(int i = 0; i < K; i++) {
		while(true)
		{
			int index_point = rand() % total_points;

			if(find(prohibited_indexes.begin(), prohibited_indexes.end(),
					index_point) == prohibited_indexes.end())
			{
				prohibited_indexes.push_back(index_point);
				h_ClusteringValues[index_point] = i;
				break;
			}
		}
	}
	
	
	// Obtener Memoria en el device
	hipMalloc((double**)&d_PointValues, numBytesPointValues); 
	
	hipMalloc((double**)&d_KCentroids, numBytesKCentroids); 
	
	hipMalloc((double**)&d_ClusteringValues, numBytesClustering); 
	
	CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 
	
	
	// Copiar datos desde el host en el device 
	hipMemcpy(d_PointValues, h_PointValues, numBytesPointValues, 
				hipMemcpyHostToDevice);
	
	hipMemcpy(d_KCentroids, h_KCentroids, numBytesKCentroids, 
				hipMemcpyHostToDevice);
	
	hipMemcpy(d_ClusteringValues, h_ClusteringValues, 
				numBytesClustering, hipMemcpyHostToDevice);
	CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
	

	// Ejecutar el kernel 
	
	nThreadsC = THREADS;
	// nBlocks = Nfil/nThreads;  // Solo funciona bien si Nfil multiplo de nThreads
	cout << "nBlocksC: " << (total_values + nThreadsC - 1)/nThreadsC << endl;
	cout << "total_values: " << total_values << endl;
	cout << "nThreadsC: " << nThreadsC << endl;
	nBlocksC = (total_values + nThreadsC - 1)/nThreadsC;  // Funciona bien en cualquier caso

	dim3 dimGridC(nBlocksC, 1, 1);
	dim3 dimBlockC(nThreadsC, K, 1);
	
	printf("\n");
	printf("Kernel de su puta madre\n");
	printf("Dimension Block: %d x %d x %d (%d) threads\n", dimBlockC.x, dimBlockC.y, dimBlockC.z, dimBlockC.x * dimBlockC.y * dimBlockC.z);
	printf("Dimension Grid: %d x %d x %d (%d) blocks\n", dimGridC.x, dimGridC.y, dimGridC.z, dimGridC.x * dimGridC.y * dimGridC.z);
  
  
	hipEventRecord(E1, 0);
	hipEventSynchronize(E1);
	
	updateCentroids<<<dimGridC,dimBlockC>>>(d_PointValues, d_KCentroids,
	  							d_ClusteringValues, total_points, total_values, K); 
	CheckCudaError((char *) "Invocar Kernel", __LINE__);
					
	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);
	/*int counter = 0;
	cudaEventRecord(E3, 0);
	cudaEventSynchronize(E3);
	bool yeray = updatePointDistances();
	cudaEventRecord(E4, 0);
	cudaEventSynchronize(E4);
	while (yeray and counter <= max_iterations) {
		++counter;
		updateCentroids(total_values);
		yeray = updatePointDistances();
	}
	cudaEventRecord(E5, 0);
	cudaEventSynchronize(E5);*/


  // Obtener el resultado desde el host 
	//cudaMemcpy(h_PointValues, d_PointValues, numBytesPointValues,
	//											cudaMemcpyDeviceToHost);
	hipMemcpy(h_KCentroids, d_KCentroids, numBytesKCentroids,
								hipMemcpyDeviceToHost);
	//cudaMemcpy(h_ClusteringValues, d_ClusteringValues, numBytesClustering,
	//							cudaMemcpyDeviceToHost); 
								
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);
  
  printClusters(h_PointValues, h_KCentroids, h_ClusteringValues);
  

  // Liberar Memoria del device 
  hipFree(d_PointValues); hipFree(d_KCentroids); 
  hipFree(d_ClusteringValues);

  hipDeviceSynchronize();
  

  hipEventElapsedTime(&TiempoUpdateCentroids, E1, E2);
  //cudaEventElapsedTime(&TiempoUpdatePointDistances, E3, E4);
  //cudaEventElapsedTime(&TiempoTotal,  E1, E5);
  
  hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
  hipEventDestroy(E4); hipEventDestroy(E5);
 
  printf("Tiempo UpdateCentroids function: %4.6f milseg\n", 
		TiempoUpdateCentroids);
  /*printf("Tiempo UpdatePointDistances function: %4.6f milseg\n", 
		TiempoUpdatePointDistances);
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);*/
  

  free(h_PointValues); free(h_KCentroids); free(h_ClusteringValues);

}


/*bool updatePointDistances(){
	double sum, min_dist;
	int min_k;
	bool change = false;
	for (int i = 0; i<PointValues.size(); ++i) {
		min_dist = 0.0;
		for (int j = 0; j<KCentroids.size(); ++j) {
			sum = 0.0;
			for (int k = 0; k<PointValues[i].size(); ++k) {
				sum += pow(KCentroids[j][k] -
					   PointValues[i][k], 2.0);
			}
			if (j == 0) {
				min_dist = sqrt(sum);
				min_k = j;
			}
			if (min_dist > sqrt(sum)) {
				min_dist = sqrt(sum);
				min_k = j;
			}
		}
		if (ClusteringValues[i] != min_k) {
			ClusteringValues[i] = min_k;
			change = true;
		}
	}
	return change;
}*/

/*void updateCentroids(double *PointValues, double *KCentroids, 
										 double *ClusteringValues){
						
	double *updatingK;
	updatingK.resize(KCentroids.size());
	for (int i = 0; i<ClusteringValues.size(); ++i) {
		vector<double> AddingK;
		for (int j = 0; j<PointValues[i].size(); ++j) {
			AddingK.push_back(PointValues[i*total_values+j]);//AddingK.push_back(PointValues[i][j]);
		}
		for (int j = 0; j<AddingK.size(); ++j) {
			updatingK[ClusteringValues[i]].push_back(AddingK[j]);
		}
	}
	vector<double> KUpdated(total_values,0);
	for (int i = 0; i<updatingK.size(); ++i) {
		vector<double> KUpdated(total_values,0);
		for (int j = 0; j<updatingK[i].size(); ++j) {
			KUpdated[j%total_values] += updatingK[i][j];
		}
		if (updatingK[i].size() > 0) {
			for (int j = 0; j<KUpdated.size(); ++j) {
				KUpdated[j] /= (updatingK[i].size()/total_values);
			}
			KCentroids[i] = KUpdated;
		}
	}
}*/


void printClusters(double *PointValues, double *KCentroids, 
									 double *ClusteringValues) {
										 
	for (int i = 0; i<K; ++i) {
		cout << "Centroid " << i << ": ";
		for (int j = 0; j<total_values; ++j) {
			int ind = i * total_values + j;
			cout << KCentroids[ind] << " ";
		}
		cout << endl;
	}
	for (int i = 0; i<total_points; ++i) {
		cout << "Point " << i << ": ";
		for (int j = 0; j<total_values; ++j) {
			int ind = i * total_values + j;
			cout << PointValues[ind] << " ";
		}
		cout << "is located on cluster: " << ClusteringValues[i] << endl;
	}
}

int error(float a, float b) {

  if (abs (a - b) / a > 0.000001) return 1;
  else  return 0;

}

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }


}


