
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <vector>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
using namespace std;

vector< vector<double> > PointValues; 
vector< vector<double> > KCentroids;
vector<int> ClusteringValues;



void printClusters();
void updateCentroids(int total_values);
bool updatePointDistances();
void CheckCudaError(char sms[], int line);

int main(int argc, char** argv) {

 
  hipEvent_t E1, E2, E3, E4, E5;
  float TiempoTotal, TiempoUpdateCentroids, TiempoUpdatePointDistances;

  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);
  hipEventCreate(&E4);
  hipEventCreate(&E5);
  
  
  int total_points, total_values, K, max_iterations;
	cin >> total_points >> total_values >> K >> max_iterations;
	if(K > total_points)
			cout << "INPUT ERROR";
			
	ClusteringValues.resize(total_points);
	for(int i = 0; i < total_points; i++) {
		vector<double> values;

		for(int j = 0; j < total_values; j++)
		{
			double value;
			cin >> value;
			values.push_back(value);
		}
		PointValues.push_back(values);
	}
	vector<int> prohibited_indexes;
	srand(1);
	for(int i = 0; i < K; i++)
	{
		while(true)
		{
			int index_point = rand() % total_points;

			if(find(prohibited_indexes.begin(), prohibited_indexes.end(),
					index_point) == prohibited_indexes.end())
			{
				prohibited_indexes.push_back(index_point);
				ClusteringValues[index_point] = i;
				break;
			}
		}
	}
	KCentroids = vector<vector<double> >(K, vector<double>(total_values));
	hipEventRecord(E1, 0);
	hipEventSynchronize(E1);
	updateCentroids(total_values); 
	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);
	
	int counter = 0;
	hipEventRecord(E3, 0);
	hipEventSynchronize(E3);
	bool yeray = updatePointDistances();
	hipEventRecord(E4, 0);
	hipEventSynchronize(E4);
	while (yeray and counter <= max_iterations) {
		++counter;
		updateCentroids(total_values);
		yeray = updatePointDistances();
	}
	cout << "LLAMADAS A UPDATECENTROIDS: " << counter << endl;
	cout << "LLAMADAS A UPDATEPOINTDISTANCES: " << counter+1 << endl;
	hipEventRecord(E5, 0);
	hipEventSynchronize(E5);


  hipEventElapsedTime(&TiempoUpdateCentroids, E1, E2);
  hipEventElapsedTime(&TiempoUpdatePointDistances, E3, E4);
  hipEventElapsedTime(&TiempoTotal,  E1, E5);

  printf("Tiempo UpdateCentroids function: %4.6f milseg\n", 
		TiempoUpdateCentroids);
  printf("Tiempo UpdatePointDistances function: %4.6f milseg\n", 
		TiempoUpdatePointDistances);
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);

  hipEventDestroy(E1); 
  hipEventDestroy(E2); hipEventDestroy(E3);
  hipEventDestroy(E4); hipEventDestroy(E5);

}

//Updatea la distancia de los puntos con las nuevas K's (si hay algun
//cambio retorna true, else false
bool updatePointDistances(){
	double sum, min_dist;
	int min_k;
	bool change = false;
	for (int i = 0; i<PointValues.size(); ++i) {
		min_dist =
		 0.0;
		for (int j = 0; j<KCentroids.size(); ++j) {
			sum = 0.0;
			for (int k = 0; k<PointValues[i].size(); ++k) {
				sum += pow(KCentroids[j][k] -
					   PointValues[i][k], 2.0);
			}
			if (j == 0) {
				min_dist = sqrt(sum);
				min_k = j;
			}
			if (min_dist > sqrt(sum)) {
				min_dist = sqrt(sum);
				min_k = j;
			}
		}
		if (ClusteringValues[i] != min_k) {
			ClusteringValues[i] = min_k;
			change = true;
		}
	}
	return change;
}

//Updatea los nuevos valores de K segun los cambios que ha habido en
//la assignacion de puntos
void updateCentroids(int total_values){
	vector<vector<double> > updatingK;
	updatingK.resize(KCentroids.size());
	for (int i = 0; i<ClusteringValues.size(); ++i) {
		vector<double> AddingK;
		for (int j = 0; j<PointValues[i].size(); ++j) {
			AddingK.push_back(PointValues[i][j]);
		}
		for (int j = 0; j<AddingK.size(); ++j) {
			updatingK[ClusteringValues[i]].push_back(AddingK[j]);
		}
	}
	vector<double> KUpdated(total_values,0);
	for (int i = 0; i<updatingK.size(); ++i) {
		vector<double> KUpdated(total_values,0);
		for (int j = 0; j<updatingK[i].size(); ++j) {
			KUpdated[j%total_values] += updatingK[i][j];
		}
		if (updatingK[i].size() > 0) {
			for (int j = 0; j<KUpdated.size(); ++j) {
				KUpdated[j] /= (updatingK[i].size()/total_values);
			}
			KCentroids[i] = KUpdated;
		}
	}
}


void printClusters() {
	for (int i = 0; i<KCentroids.size(); ++i) {
		cout << "Centroid " << i << ": ";
		for (int j = 0; j<KCentroids[i].size(); ++j) {
			cout << KCentroids[i][j] << " ";
		}
		cout << endl;
	}
	for (int i = 0; i<PointValues.size(); ++i) {
		cout << "Point " << i << ": ";
		for (int j = 0; j<PointValues[i].size(); ++j) {
			cout << PointValues[i][j] << " ";
		}
		cout << "is located on cluster: " << ClusteringValues[i] << endl;
	}
}

int error(float a, float b) {

  if (abs (a - b) / a > 0.000001) return 1;
  else  return 0;

}

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }


}


